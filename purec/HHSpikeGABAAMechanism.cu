#include "hip/hip_runtime.h"
/**
 * @file    HHNaCurrMechanism.cu
 *
 * @brief   TODO
 *
 * @details TODO
 *
 * @author  Pedro Rittner
 *
 * @date    April 23, 2014
 */
#include <stdint.h>

#include <hip/hip_runtime.h>

extern "C"
{
	#include "MyriadObject.h"
    #include "Compartment.h"
	#include "HHSomaCompartment.h"
	#include "Mechanism.h"
	#include "HHSpikeGABAAMechanism.h"
}

#include "HHSomaCompartment.cuh"
#include "HHSpikeGABAAMechanism.cuh"

__device__ __constant__ struct HHSpikeGABAAMechanism* HHSpikeGABAAMechanism_dev_t;
__device__ __constant__ struct HHSpikeGABAAMechanismClass* HHSpikeGABAAMechanismClass_dev_t;

__device__ double HHSpikeGABAAMechanism_cuda_mech_fun(void* _self,
                                                       void* pre_comp,
                                                       void* post_comp,
                                                       const double global_time,
                                                       const uint_fast32_t curr_step)
{
	struct HHSpikeGABAAMechanism* self = (struct HHSpikeGABAAMechanism*) _self;
	const struct HHSomaCompartment* c1 = (const struct HHSomaCompartment*) pre_comp;
	const struct HHSomaCompartment* c2 = (const struct HHSomaCompartment*) post_comp;

	//	Channel dynamics calculation
    const double pre_pre_vm = (curr_step > 1) ? c1->vm[curr_step-2] : INFINITY;
	const double pre_vm = c1->vm[curr_step-1];
	const double post_vm = c2->vm[curr_step-1];
    
    // If we just fired
    if (pre_vm > self->prev_vm_thresh && pre_pre_vm < self->prev_vm_thresh)
    {
        self->t_fired = global_time;
    }

    if (self->t_fired != -INFINITY)
    {
        const double g_s = expf(-(global_time - self->t_fired) / self->tau_beta) - 
            expf(-(global_time - self->t_fired) / self->tau_alpha);
        const double I_GABA = self->norm_const * -self->g_max * g_s * (post_vm - self->gaba_rev);
        return I_GABA;        
    } else {
        return 0.0;
    }
}

__device__ mech_fun_t HHSpikeGABAAMechanism_mech_fxn_t = HHSpikeGABAAMechanism_cuda_mech_fun;
