#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include <stdbool.h>
#include <string.h>
#include <math.h>
#include <dirent.h>
#include <unistd.h>
#include <pthread.h>

#ifdef CUDA
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
const bool USE_CUDA = true;
#else
const bool USE_CUDA = false;
#endif

// Myriad C API Headers
#ifdef __cplusplus
extern "C" {
#endif

#include "myriad.h"
#ifdef MYRIAD_ALLOCATOR    
#include "myriad_alloc.h"
#endif
#include "MyriadObject.h"
#include "Mechanism.h"
#include "Compartment.h"
#include "HHSomaCompartment.h"
#include "HHLeakMechanism.h"
#include "HHNaCurrMechanism.h"
#include "HHKCurrMechanism.h"
#include "HHSpikeGABAAMechanism.h"
#include "DCCurrentMech.h"
    
#ifdef __cplusplus
}
#endif

#ifdef CUDA
#include "MyriadObject.cuh"
#include "Mechanism.cuh"
#include "Compartment.cuh"
#include "HHSomaCompartment.cuh"
#include "HHLeakMechanism.cuh"
#include "HHNaCurrMechanism.cuh"
#include "HHKCurrMechanism.cuh"
#include "HHSpikeGABAAMechanism.cuh"
#include "DCCurrentMech.cuh"
#endif

////////////////
// DSAC Model //
////////////////

// Fast exponential function structure/function
#ifdef FAST_EXP
__thread union _eco _eco;
#endif

static void* new_dsac_soma(unsigned int id,
                           int_fast32_t* connect_to,
                           bool stimulate,
                           const unsigned int num_connxs)
{
	void* hh_comp_obj = myriad_new(HHSomaCompartment, id, 0, NULL, NULL, INIT_VM, CM);
	void* hh_leak_mech = myriad_new(HHLeakMechanism, id, G_LEAK, E_REV);
	void* hh_na_curr_mech = myriad_new(HHNaCurrMechanism, id, G_NA, E_NA, HH_M, HH_H);
	void* hh_k_curr_mech = myriad_new(HHKCurrMechanism, id, G_K, E_K, HH_N);

	void* dc_curr_mech = NULL;
	if (stimulate)
	{
		dc_curr_mech = myriad_new(DCCurrentMech, id, 200000, 999000, 9.0);
	} else {
		dc_curr_mech = myriad_new(DCCurrentMech, id, 200000, 999000, 0.0);
	}

	const int result =
        add_mechanism(hh_comp_obj, hh_leak_mech) ||
        add_mechanism(hh_comp_obj, hh_na_curr_mech) ||
        add_mechanism(hh_comp_obj, hh_k_curr_mech) ||
        add_mechanism(hh_comp_obj, dc_curr_mech);
    if (result)
    {
        fputs("Failed to add mechanisms to compartment", stderr);
        exit(EXIT_FAILURE);
    }

    for (uint_fast32_t i = 0; i < num_connxs; i++)
    {
        // Don't connect if it's -1
        if (connect_to[i] == -1)
        {
            continue;
        }
            
        void* hh_GABA_a_curr_mech = myriad_new(HHSpikeGABAAMechanism,
                                               connect_to[i],
                                               GABA_VM_THRESH,
                                               -INFINITY,
                                               GABA_G_MAX,
                                               GABA_TAU_ALPHA,
                                               GABA_TAU_BETA,
                                               GABA_REV);
        if (add_mechanism(hh_comp_obj, hh_GABA_a_curr_mech))
        {
            fputs("Unable to add GABA current mechanism", stderr);
            exit(EXIT_FAILURE);
        }
        DEBUG_PRINTF("GABA synapse from ID# %li -> #ID %i\n", connect_to[i], id);
    }

	return hh_comp_obj;
}

static ssize_t calc_total_size(int* num_allocs)
{
    ssize_t total_size = 0;
    
    // Class Overhead
    total_size += sizeof(struct MyriadObject) + sizeof(struct MyriadClass);
    total_size += sizeof(struct Mechanism) + sizeof(struct MechanismClass);
    total_size += sizeof(struct DCCurrentMech) + sizeof(struct DCCurrentMechClass);
    total_size += sizeof(struct HHLeakMechanism) + sizeof(struct HHLeakMechanismClass);
    total_size += sizeof(struct HHNaCurrMechanism) + sizeof(struct HHNaCurrMechanismClass);
    total_size += sizeof(struct HHKCurrMechanism) + sizeof(struct HHKCurrMechanismClass);
    total_size += sizeof(struct HHSpikeGABAAMechanism) + sizeof(struct HHSpikeGABAAMechanismClass);
    total_size += sizeof(struct Compartment) + sizeof(struct CompartmentClass);
    total_size += sizeof(struct HHSomaCompartment) + sizeof(struct HHSomaCompartmentClass);
    *num_allocs = *num_allocs + (9 * 2);

    // Objects
    total_size += sizeof(struct HHSomaCompartment) * NUM_CELLS;
    total_size += sizeof(struct DCCurrentMech) * NUM_CELLS;
    total_size += sizeof(struct HHLeakMechanism) * NUM_CELLS;
    total_size += sizeof(struct HHNaCurrMechanism) * NUM_CELLS;
    total_size += sizeof(struct HHKCurrMechanism) * NUM_CELLS;
    total_size += sizeof(struct HHSpikeGABAAMechanism) * NUM_CELLS * NUM_CELLS;
    *num_allocs = *num_allocs + (6 * NUM_CELLS) + (NUM_CELLS * NUM_CELLS);

    return total_size;
}

#if NUM_THREADS > 1

static struct _pthread_vals
{
    void** network;
    double curr_time;
    uint_fast32_t curr_step;
    uint_fast32_t num_done;
    pthread_mutex_t barrier_mutx;
    pthread_cond_t barrier_cv;
} _pthread_vals;

static inline void* _thread_run(void* arg)
{
    const int thread_id = (unsigned long int) arg;
    const int network_indx_start = thread_id * (NUM_CELLS / NUM_THREADS);
    const int network_indx_end = network_indx_start + (NUM_CELLS / NUM_THREADS) - 1;
    
    while(_pthread_vals.curr_step < SIMUL_LEN)
	{
		for (int i = network_indx_start; i < network_indx_end; i++)
		{
			simul_fxn(_pthread_vals.network[i],
                      _pthread_vals.network,
                      _pthread_vals.curr_time,
                      _pthread_vals.curr_step);
		}

        pthread_mutex_lock(&_pthread_vals.barrier_mutx);
        _pthread_vals.num_done++;
        if (_pthread_vals.num_done < NUM_THREADS)
        {
            pthread_cond_wait(&_pthread_vals.barrier_cv,
                              &_pthread_vals.barrier_mutx);
        } else {
            _pthread_vals.curr_step++;
            _pthread_vals.curr_time += DT;
            _pthread_vals.num_done = 0;
            pthread_cond_broadcast(&_pthread_vals.barrier_cv);
        }
        pthread_mutex_unlock(&_pthread_vals.barrier_mutx);
	}
    return NULL;
}
#endif /* NUM_THREADS > 1 */

// CUDA Kernel
#ifdef CUDA
__global__ void myriad_cuda_simul(void* network[NUM_CELLS])
{
    int i = threadIdx.x;
}
#endif

int main(void)
{
    srand(42);

#ifdef MYRIAD_ALLOCATOR
    int num_allocs = 0;
    const size_t total_mem_usage = calc_total_size(&num_allocs);
    if (myriad_alloc_init(total_mem_usage, num_allocs))
    {
        fputs("Unable to initialize myriad allocator\n", stderr);
        exit(EXIT_FAILURE);
    }
    DEBUG_PRINTF("total size: %lu, num allocs: %i\n", total_mem_usage, num_allocs);
    if (atexit((void (*)(void)) myriad_finalize))
    {
        fputs("Could not set myriad_finalize to run at exit\n", stderr);
        myriad_finalize();
        exit(EXIT_FAILURE);
    }
#endif /* MYRIAD_ALLOCATOR */

	initMechanism();
    initCompartment();
	initDCCurrMech();
	initHHLeakMechanism();
	initHHNaCurrMechanism();
	initHHKCurrMechanism();
	initHHSpikeGABAAMechanism();
	initHHSomaCompartment();

	void* network[NUM_CELLS] = {NULL};
    
    const uint_fast32_t num_connxs = NUM_CELLS;
    int_fast32_t to_connect[num_connxs];

	for (int_fast32_t my_id = 0; my_id < NUM_CELLS; my_id++)
	{
        memset(to_connect, 0, sizeof(int_fast32_t) * num_connxs);
        
		// All-to-All
        for (int_fast32_t j = 0; j < NUM_CELLS; j++)
        {
            if (j == my_id)
            {
                to_connect[j] = -1;  // Don't connect to ourselves
            } else {
                to_connect[j] = j;   // Connect to cell j
            }
            DEBUG_PRINTF("to_connect[%" PRIiFAST64 "]: %" PRIiFAST64 "\n", j, to_connect[j]);
        }
        
        const bool stimulate = rand() % 2 == 0;
	    network[my_id] = new_dsac_soma(my_id,
                                       to_connect,
                                       stimulate,
                                       num_connxs);
	}

#if NUM_THREADS > 1
    // Pthread parallelism
    pthread_t _threads[NUM_THREADS];

    // Initialize global pthread values
    _pthread_vals.network = network;
    _pthread_vals.curr_time = DT;
    _pthread_vals.curr_step = 1;
    _pthread_vals.num_done = 0;
    pthread_mutex_init(&_pthread_vals.barrier_mutx, NULL);
    pthread_cond_init(&_pthread_vals.barrier_cv, NULL);

    for(unsigned long int i = 0; i < NUM_THREADS; ++i)
    {
        if(pthread_create(&_threads[i], NULL, &_thread_run, (void*) i))
        {
            DEBUG_PRINTF(stderr, "Could not create thread %lu\n", i);
            exit(EXIT_FAILURE);
        }
    }
    DEBUG_PRINTF("Done creating %d threads\n", NUM_THREADS);
    for(int i = 0; i < NUM_THREADS; ++i)
    {
        if(pthread_join(_threads[i], NULL))
        {
            DEBUG_PRINTF(stderr, "Could not join thread %d\n", i);
            exit(EXIT_FAILURE);
        }
    }
#else
    double current_time = DT;
    for (uint_fast32_t curr_step = 1; curr_step < SIMUL_LEN; curr_step++)
    {
        for (uint_fast32_t i = 0; i < NUM_CELLS; i++)
        {
            simul_fxn(network[i], network, current_time, curr_step);
        }
        current_time += DT;
    }
#endif /* NUM_THREADS > 1 */

#ifdef MYRIAD_ALLOCATOR
    myriad_finalize();
#endif
    
    exit(EXIT_SUCCESS);
}
