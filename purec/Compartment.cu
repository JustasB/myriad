#include <stdint.h>
#include <hip/hip_runtime.h>

extern "C"
{
	#include "MyriadObject.h"
    #include "Compartment.h"
}

#include "MyriadObject.cuh"
#include "Compartment.cuh"

__device__ void Compartment_cuda_simul_fxn(
	void* _self,
	void** network,
    const double global_time,
	const uint_fast32_t curr_step
	)
{
	const struct Compartment* self = (const struct Compartment*) _self;
	printf("My id is %lu\n", self->id);
	// printf("My num_mechs is %u\n", self->num_mechs);
	return;
}

__device__ compartment_simul_fxn_t Compartment_cuda_compartment_fxn_t = Compartment_cuda_simul_fxn;

__device__ void cuda_simul_fxn(
	void* _self,
	void** network,
    const double global_time,
	const uint_fast32_t curr_step
	)
{
	const struct CompartmentClass* m_class =
		(const struct CompartmentClass*) cuda_myriad_class_of((void*) _self);

	return m_class->m_compartment_simul_fxn(_self, network, global_time, curr_step);
}

__device__ __constant__ struct Compartment* Compartment_dev_t;
__device__ __constant__ struct CompartmentClass* CompartmentClass_dev_t;
