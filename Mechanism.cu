#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>

#include <hip/hip_runtime.h>

extern "C"
{
    #include "MyriadObject.h"
    #include "Mechanism.h"
}

#include "MyriadObject.cuh"
#include "Mechanism.cuh"


__device__ double Mechanism_cuda_mechanism_fxn(void* _self,
                                               void* pre_comp,
                                               void* post_comp,
                                               const double global_time,
                                               const uint64_t curr_step)
{
	// const struct Mechanism* self = (const struct Mechanism*) _self;
	// printf("My source id is %u\n", self->source_id);
	return 0.0;
}

__device__ mech_fun_t Mechanism_cuda_mechanism_fxn_t = Mechanism_cuda_mechanism_fxn;

__device__ double cuda_mechanism_fxn(void* _self,
                                     void* pre_comp,
                                     void* post_comp,
                                     const double global_time,
                                     const uint64_t curr_step)
{
	const struct MechanismClass* m_class = (const struct MechanismClass*) cuda_myriad_class_of(_self);

	return m_class->m_mech_fxn(_self, pre_comp, post_comp, global_time, curr_step);
}

__device__ __constant__ struct Mechanism* Mechanism_dev_t = NULL;
__device__ __constant__ struct MechanismClass* MechanismClass_dev_t = NULL;
