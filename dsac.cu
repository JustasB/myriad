#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <math.h>

#ifdef CUDA
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#endif

// Myriad C API Headers
extern "C"
{
	#include "myriad_debug.h"
    #include "MyriadObject.h"
	#include "Mechanism.h"
	#include "Compartment.h"
	#include "HHSomaCompartment.h"
	#include "HHLeakMechanism.h"
	#include "HHNaCurrMechanism.h"
	#include "HHKCurrMechanism.h"
//    #include "HHGradedGABAAMechanism.h"
    #include "HHSpikeGABAAMechanism.h"
    #include "DCCurrentMech.h"
}

#ifdef CUDA
#include "myriad_debug.h"
#include "MyriadObject.cuh"
#include "Mechanism.cuh"
#include "Compartment.cuh"
#endif

////////////////
// DSAC Model //
////////////////

// Simulation parameters
#define SIMUL_LEN 1000 
#define DT 0.001
#define NUM_CELLS 2
// Leak params
#define G_LEAK 1.0
#define E_REV -65.0
// Sodium params
#define G_NA 35.0
#define E_NA 55.0
#define HH_M 0.5
#define HH_H 0.1
// Potassium params
#define G_K 9.0
#define E_K -90.0
#define HH_N 0.1
// Compartment Params
#define CM 1.0
#define INIT_VM -65.0
// GABA-a Params
#define GABA_VM_THRESH 0.0
#define GABA_G_MAX 0.1
#define GABA_TAU_ALPHA 0.08333333333333333
#define GABA_TAU_BETA 10.0
#define GABA_REV -75.0

#ifdef CUDA
__global__ void cuda_hh_compartment_test(void** network)
{
	struct HHSomaCompartment* curr_comp = (struct HHSomaCompartment*) network[threadIdx.x];

	double curr_time = DT;
    unsigned int curr_step = 1;

	while (curr_step < SIMUL_LEN)
	{
		cuda_simul_fxn(curr_comp, network, DT, curr_time, curr_step);
        
        if (threadIdx.x == 0)
        {
            curr_time += DT;
            curr_step++;
        }
        __syncthreads();
	}
}
#endif

static void* new_dsac_soma(unsigned int id, unsigned int* connect_to, const unsigned int num_connxs)
{
	void* hh_comp_obj = myriad_new(HHSomaCompartment, id, 0, NULL, SIMUL_LEN, NULL, INIT_VM, CM);

	void* hh_leak_mech = myriad_new(HHLeakMechanism, id, G_LEAK, E_REV);
	void* hh_na_curr_mech = myriad_new(HHNaCurrMechanism, id, G_NA, E_NA, HH_M, HH_H);
	void* hh_k_curr_mech = myriad_new(HHKCurrMechanism, id, G_K, E_K, HH_N);
	void* dc_curr_mech = NULL;

	if (id == 0)
	{
		dc_curr_mech = myriad_new(DCCurrentMech, id, 200000, 999000, 9.0);
	} else {
		dc_curr_mech = myriad_new(DCCurrentMech, id, 200000, 999000, 0.0);
	}

    #ifdef CUDA
    assert(EXIT_SUCCESS == add_mechanism(hh_comp_obj, myriad_cudafy(hh_leak_mech, 0)));
    assert(EXIT_SUCCESS == add_mechanism(hh_comp_obj, myriad_cudafy(hh_na_curr_mech, 0)));
    assert(EXIT_SUCCESS == add_mechanism(hh_comp_obj, myriad_cudafy(hh_k_curr_mech, 0)));
    assert(EXIT_SUCCESS == add_mechanism(hh_comp_obj, myriad_cudafy(dc_curr_mech, 0)));
    #else
	assert(EXIT_SUCCESS == add_mechanism(hh_comp_obj, hh_leak_mech));
	assert(EXIT_SUCCESS == add_mechanism(hh_comp_obj, hh_na_curr_mech));
	assert(EXIT_SUCCESS == add_mechanism(hh_comp_obj, hh_k_curr_mech));
	assert(EXIT_SUCCESS == add_mechanism(hh_comp_obj, dc_curr_mech));
    #endif

	if (num_connxs > 0)
	{
		for (unsigned int i = 0; i < num_connxs; i++)
		{
			void* hh_GABA_a_curr_mech = 
				myriad_new
				(
					HHSpikeGABAAMechanism,
					connect_to[i],
                    GABA_VM_THRESH,
                    -INFINITY,
                    GABA_G_MAX,
                    GABA_TAU_ALPHA,
                    GABA_TAU_BETA,
                    GABA_REV
				);

            #ifdef CUDA
            assert(EXIT_SUCCESS == add_mechanism(hh_comp_obj, myriad_cudafy(hh_GABA_a_curr_mech, 0)));
            #else
			assert(EXIT_SUCCESS == add_mechanism(hh_comp_obj, hh_GABA_a_curr_mech));
            #endif

			printf("Made GABA synapse starting at cell %i ending at cell %i\n", connect_to[i], id);
		}
	}
    
    #ifdef CUDA
    return myriad_cudafy(hh_comp_obj, 0);
    #else
	return hh_comp_obj;
    #endif
}

static int dsac()
{
    int cuda_init = 0;
    #ifdef CUDA
    cuda_init = 1;
    #endif

	initMechanism(cuda_init);
	initDCCurrMech(cuda_init);
	initHHLeakMechanism(cuda_init);
	initHHNaCurrMechanism(cuda_init);
	initHHKCurrMechanism(cuda_init);
	initHHSpikeGABAAMechanism(cuda_init);
	initCompartment(cuda_init);
	initHHSomaCompartment(cuda_init);

	void** network = (void**) calloc(NUM_CELLS, sizeof(void*));

	for (int i = 0; i < NUM_CELLS; i++)
	{
		//TODO: Guarantee % connectivity b/w cells in network
		const unsigned int num_connxs = 1;
		unsigned int* to_connect = (unsigned int*) calloc(num_connxs, sizeof(unsigned int));
		
		//TODO: Get rid of this hack
		if (i == 0)
		{
			to_connect[0] = 1;
		} else if (i == 1) {
			to_connect[0] = 0;
		}

	    network[i] = new_dsac_soma(i, to_connect, 1);
	}

    puts("Starting simulation...");

    #ifdef CUDA
    void** cuda_network = NULL;
    CUDA_CHECK_RETURN(hipMalloc(&cuda_network, sizeof(void*) * NUM_CELLS));
    CUDA_CHECK_RETURN(hipMemcpy(cuda_network, network, sizeof(void*) * NUM_CELLS, hipMemcpyHostToDevice));
    cuda_hh_compartment_test<<<1, NUM_CELLS>>>(cuda_network);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    #else
	double curr_time = DT;
	for (unsigned int curr_step = 1; curr_step < SIMUL_LEN; curr_step++)
	{
		for (int i = 0; i < NUM_CELLS; i++)
		{
			simul_fxn(network[i], network, DT, curr_time, curr_step);
		}
		curr_time += DT;
	}
    #endif

    puts("Simulation completed successfully.");

    /*
	for (int i = 0; i < NUM_CELLS; i++)
	{
		struct HHSomaCompartment* curr_comp = (struct HHSomaCompartment*) network[i];
		char* fname = (char*) malloc(sizeof("cell0.dat"));
		sprintf(fname, "cell%i.dat", i);
		FILE* p_file = fopen(fname,"wb");
		fwrite(curr_comp->soma_vm, sizeof(double), curr_comp->soma_vm_len, p_file);
		fclose(p_file);
	}
    */

    #ifdef CUDA
    CUDA_CHECK_RETURN(hipDeviceReset());
    #endif

    return EXIT_SUCCESS;
}

///////////////////
// Main function //
///////////////////
int main(int argc, char const *argv[])
{
    puts("Hello World!\n");

	assert(EXIT_SUCCESS == dsac());

    puts("\nDone.");

    return EXIT_SUCCESS;
}
