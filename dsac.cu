#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include <stdbool.h>
#include <assert.h>
#include <string.h>
#include <math.h>
#include <dirent.h>
#include <unistd.h>

#ifdef CUDA
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#endif

// Myriad C API Headers
extern "C"
{
    #include "myriad.h"
    #include "MyriadObject.h"
	#include "Mechanism.h"
	#include "Compartment.h"
	#include "HHSomaCompartment.h"
	#include "HHLeakMechanism.h"
	#include "HHNaCurrMechanism.h"
	#include "HHKCurrMechanism.h"
//    #include "HHGradedGABAAMechanism.h"
    #include "HHSpikeGABAAMechanism.h"
    #include "DCCurrentMech.h"
}

#ifdef CUDA
#include "MyriadObject.cuh"
#include "Mechanism.cuh"
#include "Compartment.cuh"
#endif


////////////////
// DSAC Model //
////////////////


#ifdef CUDA
__global__ void cuda_hh_compartment_test(void* hh_comp_obj, void* network)
{
	void* dev_arr[1];
	dev_arr[0] = network;

	struct HHSomaCompartment* curr_comp = (struct HHSomaCompartment*) hh_comp_obj;

	double curr_time = DT;
	for (unsigned int curr_step = 1; curr_step < SIMUL_LEN; curr_step++)
	{
		cuda_simul_fxn(curr_comp, (void**) dev_arr, DT, curr_time, curr_step);
		curr_time += DT;
	}
}
#endif

static void* new_dsac_soma(unsigned int id,
                           int64_t* connect_to,
                           bool stimulate,
                           const unsigned int num_connxs)
{
	void* hh_comp_obj = myriad_new(HHSomaCompartment, id, 0, NULL, NULL, INIT_VM, CM);
	void* hh_leak_mech = myriad_new(HHLeakMechanism, id, G_LEAK, E_REV);
	void* hh_na_curr_mech = myriad_new(HHNaCurrMechanism, id, G_NA, E_NA, HH_M, HH_H);
	void* hh_k_curr_mech = myriad_new(HHKCurrMechanism, id, G_K, E_K, HH_N);

	void* dc_curr_mech = NULL;
	if (stimulate)
	{
		dc_curr_mech = myriad_new(DCCurrentMech, id, 200000, 999000, 9.0);
	} else {
		dc_curr_mech = myriad_new(DCCurrentMech, id, 200000, 999000, 0.0);
	}

	assert(0 == add_mechanism(hh_comp_obj, hh_leak_mech));
	assert(0 == add_mechanism(hh_comp_obj, hh_na_curr_mech));
	assert(0 == add_mechanism(hh_comp_obj, hh_k_curr_mech));
	assert(0 == add_mechanism(hh_comp_obj, dc_curr_mech));

    for (uint64_t i = 0; i < num_connxs; i++)
    {
        // Don't connect if it's -1
        if (connect_to[i] == -1)
        {
            continue;
        }
            
        void* hh_GABA_a_curr_mech = myriad_new(HHSpikeGABAAMechanism,
                                               connect_to[i],
                                               GABA_VM_THRESH,
                                               -INFINITY,
                                               GABA_G_MAX,
                                               GABA_TAU_ALPHA,
                                               GABA_TAU_BETA,
                                               GABA_REV);
        assert(0 == add_mechanism(hh_comp_obj, hh_GABA_a_curr_mech));
        printf("GABA synapse from ID# %" PRIi64 " -> #ID %i\n",
               connect_to[i],
               id);
    }

	return hh_comp_obj;
}

static ssize_t calc_total_size(int* num_allocs)
{
    ssize_t total_size = 0;
    
    // Scan module (read: current) directory for all C files
    /*
    int num_modules = 0;
    DIR* dp = opendir("./");  //TODO: Change this to look in module directory
    if (dp != NULL)
    {
        struct dirent *ep;
        while ((ep = readdir(dp)))
        {
            // Check if file is a C file
            if (strstr(ep->d_name, ".c") != NULL)
            {
                num_modules++;
            }
        }
        assert(closedir(dp) == 0);
    } else {
        perror("calc_total_size: Couldn't open the current directory");
        return -1;
    }
    */

    // Class Overhead
    total_size += sizeof(struct MyriadObject) + sizeof(struct MyriadClass);
    total_size += sizeof(struct Mechanism) + sizeof(struct MechanismClass);
    total_size += sizeof(struct DCCurrentMech) + sizeof(struct DCCurrentMechClass);
    total_size += sizeof(struct HHLeakMechanism) + sizeof(struct HHLeakMechanismClass);
    total_size += sizeof(struct HHNaCurrMechanism) + sizeof(struct HHNaCurrMechanismClass);
    total_size += sizeof(struct HHKCurrMechanism) + sizeof(struct HHKCurrMechanismClass);
    total_size += sizeof(struct HHSpikeGABAAMechanism) + sizeof(struct HHSpikeGABAAMechanismClass);
    total_size += sizeof(struct Compartment) + sizeof(struct CompartmentClass);
    total_size += sizeof(struct HHSomaCompartment) + sizeof(struct HHSomaCompartmentClass);
    *num_allocs = *num_allocs + (9 * 2);

    // Actual usage
    total_size += sizeof(struct HHSomaCompartment) * NUM_CELLS;
    total_size += sizeof(struct DCCurrentMech) * NUM_CELLS;
    total_size += sizeof(struct HHLeakMechanism) * NUM_CELLS;
    total_size += sizeof(struct HHNaCurrMechanism) * NUM_CELLS;
    total_size += sizeof(struct HHKCurrMechanism) * NUM_CELLS;
    total_size += sizeof(struct HHSpikeGABAAMechanism) * NUM_CELLS * NUM_CELLS;
    *num_allocs = *num_allocs + (6 * NUM_CELLS) + (NUM_CELLS * NUM_CELLS);
    
    return total_size;
}


static int dsac()
{
    #ifdef MYRIAD_ALLOCATOR
    int num_allocs = 0;
    const size_t total_mem_usage = calc_total_size(&num_allocs);
    assert(myriad_alloc_init(total_mem_usage, num_allocs) == 0);
    printf("total size: %lu, num allocs: %i\n", total_mem_usage, num_allocs);
    #endif

	initMechanism(false);
	initDCCurrMech(false);
	initHHLeakMechanism(false);
	initHHNaCurrMechanism(false);
	initHHKCurrMechanism(false);
	initHHSpikeGABAAMechanism(false);
	initCompartment(false);
	initHHSomaCompartment(false);

	void* network[NUM_CELLS];
    // memset(network, 0, sizeof(void*) * NUM_CELLS);  // Necessary?
    
    const unsigned int num_connxs = NUM_CELLS;
    int64_t to_connect[num_connxs];

	for (unsigned int my_id = 0; my_id < NUM_CELLS; my_id++)
	{
        memset(to_connect, 0, sizeof(int64_t) * num_connxs);
        
		// All-to-All
        for (int64_t j = 0; j < NUM_CELLS; j++)
        {
            if (j == my_id)
            {
                to_connect[j] = -1;  // Don't connect to ourselves
            } else {
                to_connect[j] = j;   // Connect to cell j
            }
            printf("to_connect[%" PRIi64 "]: %" PRIi64 "\n", j, to_connect[j]);
        }
        
        const bool stimulate = rand() % 2 == 0;
	    network[my_id] = new_dsac_soma(my_id,
                                       to_connect,
                                       stimulate,
                                       num_connxs);
	}

    // Run simulation
	double curr_time = DT;
	for (unsigned int curr_step = 1; curr_step < SIMUL_LEN; curr_step++)
	{
#pragma GCC ivdep
		for (int i = 0; i < NUM_CELLS; i++)
		{
			simul_fxn(network[i], network, DT, curr_time, curr_step);
		}
		curr_time += DT;
	}

    // Cleanup
    /* 
- Refactored all instances of "unsigned int" to uint64_t
- Changed cuda_init for initializers to use bool instead of int
- Added initial support for native myriad allocator
     */
    assert(myriad_finalize() == 0);

    return 0;
}

///////////////////
// Main function //
///////////////////
int main(int argc, char const *argv[])
{
    srand(42);
    puts("Hello World!\n");

	assert(0 == dsac());

    puts("\nDone.");

    return EXIT_SUCCESS;
}
