#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include <stdbool.h>
#include <assert.h>
#include <string.h>
#include <math.h>
#include <dirent.h>
#include <unistd.h>
#include <pthread.h>

#ifdef CUDA
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#endif

// Myriad C API Headers
extern "C"
{
    #include "myriad.h"
    #include "MyriadObject.h"
	#include "Mechanism.h"
	#include "Compartment.h"
	#include "HHSomaCompartment.h"
	#include "HHLeakMechanism.h"
	#include "HHNaCurrMechanism.h"
	#include "HHKCurrMechanism.h"
//    #include "HHGradedGABAAMechanism.h"
    #include "HHSpikeGABAAMechanism.h"
    #include "DCCurrentMech.h"
}

#ifdef CUDA
#include "MyriadObject.cuh"
#include "Mechanism.cuh"
#include "Compartment.cuh"
#endif

////////////////
// DSAC Model //
////////////////

// Fast exponential function structure/function
#ifdef FAST_EXP
__thread union _eco _eco;
#ifdef USE_DDTABLE
double _exp(double y)
{
    _eco.n.i = EXP_A * (y) + (1072693248 - EXP_C);
    return _eco.d;
}
#endif
#endif

static void* new_dsac_soma(unsigned int id,
                           int64_t* connect_to,
                           bool stimulate,
                           const unsigned int num_connxs)
{
	void* hh_comp_obj = myriad_new(HHSomaCompartment, id, 0, NULL, NULL, INIT_VM, CM);
	void* hh_leak_mech = myriad_new(HHLeakMechanism, id, G_LEAK, E_REV);
	void* hh_na_curr_mech = myriad_new(HHNaCurrMechanism, id, G_NA, E_NA, HH_M, HH_H);
	void* hh_k_curr_mech = myriad_new(HHKCurrMechanism, id, G_K, E_K, HH_N);

	void* dc_curr_mech = NULL;
	if (stimulate)
	{
		dc_curr_mech = myriad_new(DCCurrentMech, id, 200000, 999000, 9.0);
	} else {
		dc_curr_mech = myriad_new(DCCurrentMech, id, 200000, 999000, 0.0);
	}

	assert(0 == add_mechanism(hh_comp_obj, hh_leak_mech));
	assert(0 == add_mechanism(hh_comp_obj, hh_na_curr_mech));
	assert(0 == add_mechanism(hh_comp_obj, hh_k_curr_mech));
	assert(0 == add_mechanism(hh_comp_obj, dc_curr_mech));

    for (uint64_t i = 0; i < num_connxs; i++)
    {
        // Don't connect if it's -1
        if (connect_to[i] == -1)
        {
            continue;
        }
            
        void* hh_GABA_a_curr_mech = myriad_new(HHSpikeGABAAMechanism,
                                               connect_to[i],
                                               GABA_VM_THRESH,
                                               -INFINITY,
                                               GABA_G_MAX,
                                               GABA_TAU_ALPHA,
                                               GABA_TAU_BETA,
                                               GABA_REV);
        assert(0 == add_mechanism(hh_comp_obj, hh_GABA_a_curr_mech));
        DEBUG_PRINTF("GABA synapse from ID# %" PRIi64 " -> #ID %i\n",
                     connect_to[i],
                     id);
    }

	return hh_comp_obj;
}

#ifndef MYRIAD_ALLOCATOR
static ssize_t calc_total_size(int* num_allocs) __attribute__((unused));
#endif
static ssize_t calc_total_size(int* num_allocs)
{
    ssize_t total_size = 0;
    
    // Scan module (read: current) directory for all C files
    /*
    int num_modules = 0;
    DIR* dp = opendir("./");  //TODO: Change this to look in module directory
    if (dp != NULL)
    {
        struct dirent *ep;
        while ((ep = readdir(dp)))
        {
            // Check if file is a C file
            if (strstr(ep->d_name, ".c") != NULL)
            {
                num_modules++;
            }
        }
        assert(closedir(dp) == 0);
    } else {
        perror("calc_total_size: Couldn't open the current directory");
        return -1;
    }
    */

    // Class Overhead
    total_size += sizeof(struct MyriadObject) + sizeof(struct MyriadClass);
    total_size += sizeof(struct Mechanism) + sizeof(struct MechanismClass);
    total_size += sizeof(struct DCCurrentMech) + sizeof(struct DCCurrentMechClass);
    total_size += sizeof(struct HHLeakMechanism) + sizeof(struct HHLeakMechanismClass);
    total_size += sizeof(struct HHNaCurrMechanism) + sizeof(struct HHNaCurrMechanismClass);
    total_size += sizeof(struct HHKCurrMechanism) + sizeof(struct HHKCurrMechanismClass);
    total_size += sizeof(struct HHSpikeGABAAMechanism) + sizeof(struct HHSpikeGABAAMechanismClass);
    total_size += sizeof(struct Compartment) + sizeof(struct CompartmentClass);
    total_size += sizeof(struct HHSomaCompartment) + sizeof(struct HHSomaCompartmentClass);
    *num_allocs = *num_allocs + (9 * 2);

    // Objects
    total_size += sizeof(struct HHSomaCompartment) * NUM_CELLS;
    total_size += sizeof(struct DCCurrentMech) * NUM_CELLS;
    total_size += sizeof(struct HHLeakMechanism) * NUM_CELLS;
    total_size += sizeof(struct HHNaCurrMechanism) * NUM_CELLS;
    total_size += sizeof(struct HHKCurrMechanism) * NUM_CELLS;
    total_size += sizeof(struct HHSpikeGABAAMechanism) * NUM_CELLS * NUM_CELLS;
    *num_allocs = *num_allocs + (6 * NUM_CELLS) + (NUM_CELLS * NUM_CELLS);

    // DDTABLE
    #ifdef USE_DDTABLE
    *num_allocs = *num_allocs + 1;
    total_size += sizeof(struct ddtable);
    total_size += sizeof(int_fast8_t) * DDTABLE_NUM_KEYS;
    total_size += 2 * sizeof(double) * DDTABLE_NUM_KEYS;
    #endif
    
    return total_size;
}

#ifdef USE_DDTABLE
ddtable_t exp_table = NULL;
#endif /* USE_DDTABLE */

#if NUM_THREADS > 1
struct _pthread_vals
{
    void** network;
    double curr_time;
    uint64_t curr_step;
    uint_fast32_t num_done;
    pthread_mutex_t barrier_mutx;
    pthread_cond_t barrier_cv;
} _pthread_vals;

static inline void* _thread_run(void* arg)
{
    const int thread_id = (unsigned long int) arg;
    const int network_indx_start = thread_id * (NUM_CELLS / NUM_THREADS);
    const int network_indx_end = network_indx_start + (NUM_CELLS / NUM_THREADS) - 1;
    
    while(_pthread_vals.curr_step < SIMUL_LEN)
	{
#pragma GCC ivdep
		for (int i = network_indx_start; i < network_indx_end; i++)
		{
			simul_fxn(_pthread_vals.network[i],
                      _pthread_vals.network,
                      _pthread_vals.curr_time,
                      _pthread_vals.curr_step);
		}

        pthread_mutex_lock(&_pthread_vals.barrier_mutx);
        _pthread_vals.num_done++;
        if (_pthread_vals.num_done < NUM_THREADS)
        {
            pthread_cond_wait(&_pthread_vals.barrier_cv,
                              &_pthread_vals.barrier_mutx);
        } else {
            _pthread_vals.curr_step++;
            _pthread_vals.curr_time += DT;
            _pthread_vals.num_done = 0;
            pthread_cond_broadcast(&_pthread_vals.barrier_cv);
        }
        pthread_mutex_unlock(&_pthread_vals.barrier_mutx);
	}

    return NULL;
}
#endif /* NUM_THREADS > 1 */

static int dsac()
{
#ifdef MYRIAD_ALLOCATOR
    int num_allocs = 0;
    const size_t total_mem_usage = calc_total_size(&num_allocs);
    assert(myriad_alloc_init(total_mem_usage, num_allocs) == 0);
    DEBUG_PRINTF("total size: %lu, num allocs: %i\n", total_mem_usage, num_allocs);
#endif /* MYRIAD_ALLOCATOR */

#ifdef USE_DDTABLE
    exp_table = ddtable_new(DDTABLE_NUM_KEYS);
#endif /* USE_DDTABLE */

	initMechanism(false);
	initDCCurrMech(false);
	initHHLeakMechanism(false);
	initHHNaCurrMechanism(false);
	initHHKCurrMechanism(false);
	initHHSpikeGABAAMechanism(false);
	initCompartment(false);
	initHHSomaCompartment(false);

	void* network[NUM_CELLS];
    // memset(network, 0, sizeof(void*) * NUM_CELLS);  // Necessary?
    
    const unsigned int num_connxs = NUM_CELLS;
    int64_t to_connect[num_connxs];

	for (unsigned int my_id = 0; my_id < NUM_CELLS; my_id++)
	{
        memset(to_connect, 0, sizeof(int64_t) * num_connxs);
        
		// All-to-All
        for (int64_t j = 0; j < NUM_CELLS; j++)
        {
            if (j == my_id)
            {
                to_connect[j] = -1;  // Don't connect to ourselves
            } else {
                to_connect[j] = j;   // Connect to cell j
            }
            DEBUG_PRINTF("to_connect[%" PRIi64 "]: %" PRIi64 "\n", j, to_connect[j]);
        }
        
        const bool stimulate = rand() % 2 == 0;
	    network[my_id] = new_dsac_soma(my_id,
                                       to_connect,
                                       stimulate,
                                       num_connxs);
	}

#if NUM_THREADS > 1
    // Pthread parallelism
    pthread_t _threads[NUM_THREADS];

    // Initialize global pthread values
    _pthread_vals.network = network;
    _pthread_vals.curr_time = DT;
    _pthread_vals.curr_step = 1;
    _pthread_vals.num_done = 0;
    pthread_mutex_init(&_pthread_vals.barrier_mutx, NULL);
    pthread_cond_init(&_pthread_vals.barrier_cv, NULL);

    for(unsigned long int i = 0; i < NUM_THREADS; ++i)
    {
        if(pthread_create(&_threads[i], NULL, &_thread_run, (void*) i))
        {
            fprintf(stderr, "Could not create thread %lu\n", i);
            return -1;
        }
    }
    for(int i = 0; i < NUM_THREADS; ++i)
    {
        if(pthread_join(_threads[i], NULL))
        {
            fprintf(stderr, "Could not join thread %d\n", i);
            return -1;
        }
    }
#else
    double current_time = DT;
    for (uint_fast64_t curr_step = 1; curr_step < SIMUL_LEN; curr_step++)
    {
#pragma GCC ivdep
        for (uint_fast64_t i = 0; i < NUM_CELLS; i++)
        {
            simul_fxn(network[i], network, current_time, curr_step);
        }
        current_time += DT;
    }
#endif /* NUM_THREADS > 1 */

    // Cleanup
    #ifdef USE_DDTABLE
    ddtable_free(exp_table);
    #endif
    
    #ifdef MYRIAD_ALLOCATOR
    assert(myriad_finalize() == 0);
    #endif
    
    return 0;
}

///////////////////
// Main function //
///////////////////
int main()
{
    srand(42);
    // puts("Hello World!\n");

	assert(0 == dsac());

    // puts("\nDone.");

    return EXIT_SUCCESS;
}
