#include "hip/hip_runtime.h"
/**
   @file    HHSpikeGABAACurrMechanism.cu
 
   @brief   Hodgkin-Huxley Spike-mediated GABA-a Mechanism CUDA implementation file.
 
   @details Defines the Hodgkin-Huxley Spike-mediated GABA-a Mechanism CUDA implementation for Myriad
 
   @author  Pedro Rittner
 
   @date    June 2, 2014
 */
#include <stdio.h>

#include <hip/hip_runtime.h>

extern "C"
{
    #include "myriad_debug.h"
	#include "MyriadObject.h"
    #include "Compartment.h"
	#include "HHSomaCompartment.h"
	#include "Mechanism.h"
	#include "HHSpikeGABAAMechanism.h"
}

#include "HHSomaCompartment.cuh"
#include "HHSpikeGABAAMechanism.cuh"

__device__ __constant__ struct HHSpikeGABAAMechanism* HHSpikeGABAAMechanism_dev_t;
__device__ __constant__ struct HHSpikeGABAAMechanismClass* HHSpikeGABAAMechanismClass_dev_t;

__device__ double HHSpikeGABAAMechanism_cuda_mech_fun(
    void* _self,
	void* pre_comp,
	void* post_comp,
	const double dt,
	const double global_time,
	const unsigned int curr_step
	)
{
	struct HHSpikeGABAAMechanism* self = (struct HHSpikeGABAAMechanism*) _self;
	const struct HHSomaCompartment* c1 = (const struct HHSomaCompartment*) pre_comp;
	const struct HHSomaCompartment* c2 = (const struct HHSomaCompartment*) post_comp;

	//	Channel dynamics calculation
    const double pre_pre_vm = (curr_step > 1) ? c1->soma_vm[curr_step-2] : INFINITY;
	const double pre_vm = c1->soma_vm[curr_step-1];
	const double post_vm = c2->soma_vm[curr_step-1];
    
    // If we just fired
    if (pre_vm > self->prev_vm_thresh && pre_pre_vm < self->prev_vm_thresh)
    {
        self->t_fired = global_time;
    }

    if (self->t_fired != -INFINITY)
    {
        const double g_s = exp(-(global_time - self->t_fired) / self->tau_beta) - 
            exp(-(global_time - self->t_fired) / self->tau_alpha);
        const double I_GABA = self->norm_const * -self->g_max * g_s * (post_vm - self->gaba_rev);
        return I_GABA;        
    } else {
        return 0.0;
    }
}

__device__ mech_fun_t HHSpikeGABAAMechanism_mech_fxn_t = HHSpikeGABAAMechanism_cuda_mech_fun;
