#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>

#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Myriad C API Headers
extern "C"
{
	#include "myriad_debug.h"
    #include "MyriadObject.h"
	#include "Mechanism.h"
	#include "Compartment.h"
	#include "HHSomaCompartment.h"
}

#include "MyriadObject.cuh"
#include "Mechanism.cuh"
#include "Compartment.cuh"

///////////////////
// Test CUDA OOP //
///////////////////

__global__ void cuda_oop_test(void* c_obj)
{
    printf("\tsize(GPU): %lu\n", cuda_myriad_size_of(c_obj));
    printf("\tis_a: %s\n", cuda_myriad_is_a(c_obj, MyriadObject_dev_t) ? "TRUE" : "FALSE");
    printf("\tis_of: %s\n", cuda_myriad_is_of(c_obj, MyriadObject_dev_t) ? "TRUE" : "FALSE");
}

static int cuda_oop()
{
    initCUDAObjects();
    
    void* my_obj = myriad_new(MyriadObject);
	assert(my_obj);
    
    void* my_cuda_obj = myriad_cudafy(my_obj, 0);

    // BLAH
    const int nThreads = 1; // NUM_CUDA_THREADS;
    const int nBlocks = 1;

    dim3 dimGrid(nBlocks);
    dim3 dimBlock(nThreads);

    // Test
    #ifndef __clang__
    cuda_oop_test<<<dimGrid, dimBlock>>>(my_cuda_obj); // Not an error
    #endif
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());
	
	printf("\tCPU Size (again): %lu\n", myriad_size_of(my_obj));

	// Free
	CUDA_CHECK_RETURN(hipFree(my_cuda_obj));
    assert(myriad_dtor(my_obj) == EXIT_SUCCESS);

    hipDeviceReset();

    return EXIT_SUCCESS;
}

////////////////////
// Test Mechanism //
////////////////////

__global__ void cuda_mechansim_test(void* obj)
{
	struct Mechanism* self = (struct Mechanism*) obj;
	struct MechanismClass* self_c = (struct MechanismClass*) cuda_myriad_class_of(self);
	printf("\tMy ptr: %p\n", self);
	printf("\tMy ID: %i\n", self->source_id);
	printf("\tMy class: %p\n", self->_.m_class);
	printf("\tGPU, my size: %lu\n", cuda_myriad_size_of(obj));
	printf("\tMechanism fxn: %p\n", self_c->m_mech_fxn);
	printf("\tMechanism fxn invocation: %f\n", self_c->m_mech_fxn(self, NULL, NULL, 0.0, 0.0, 0));
	printf("\tMechanism fxn indirect call: %f\n", cuda_mechanism_fxn(self, NULL, NULL, 0.0, 0.0, 0));
}

static int mechanism_test()
{
	initCUDAObjects();
	initMechanism(1);

	void* mech_obj = NULL, *dev_mech_obj = NULL;

	mech_obj = myriad_new(Mechanism, 1);

	UNIT_TEST_VAL_EQ(myriad_size_of(mech_obj), sizeof(struct Mechanism));

	mechanism_fxn(mech_obj, NULL, NULL, 0, 0, 0);

	dev_mech_obj = myriad_cudafy(mech_obj, 0);

    // BLAH
    const int nThreads = 1; // NUM_CUDA_THREADS;
    const int nBlocks = 1;

    dim3 dimGrid(nBlocks);
    dim3 dimBlock(nThreads);

    // Test
    #ifndef __clang__
    cuda_mechansim_test<<<dimGrid, dimBlock>>>(dev_mech_obj); // Not an error
    #endif
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());

	// Free
	CUDA_CHECK_RETURN(hipFree(dev_mech_obj));
	assert(!myriad_dtor(mech_obj));

    hipDeviceReset();

    return EXIT_SUCCESS;
}

//////////////////////
// Test Compartment //
//////////////////////

__global__ void cuda_compartment_test(void* obj)
{
	struct Compartment* self = (struct Compartment*) obj;
	struct CompartmentClass* self_c = (struct CompartmentClass*) cuda_myriad_class_of(self);
	printf("\tMy ptr: %p\n", self);
	printf("\tMy ID: %i\n", self->id);
	printf("\tMy class: %p\n", self->_.m_class);
	printf("\tGPU, my size: %lu\n", cuda_myriad_size_of(obj));
	printf("\tCompartment fxn: %p\n", self_c->m_comp_fxn);
	printf("\tCompartment fxn invocation: "); self_c->m_comp_fxn(self, NULL, 0.0, 0.0, 0);
	printf("\tCompartent fxn indirect call: "); cuda_simul_fxn(self, NULL, 0.0, 0.0, 0);
}

static int compartment_test()
{
	initCUDAObjects();
	initCompartment(1);

	void* comp_obj = NULL, *dev_comp_obj = NULL;

	comp_obj = myriad_new(Compartment, 5, 42, NULL);

	UNIT_TEST_VAL_EQ(myriad_size_of(comp_obj), sizeof(struct Compartment));

	simul_fxn(comp_obj, NULL, 0.0, 0.0, 0);

	dev_comp_obj = myriad_cudafy(comp_obj, 0);

    // BLAH
    const int nThreads = 1; // NUM_CUDA_THREADS;
    const int nBlocks = 1;

    dim3 dimGrid(nBlocks);
    dim3 dimBlock(nThreads);

    // Test
    #ifndef __clang__
    cuda_compartment_test<<<dimGrid, dimBlock>>>(dev_comp_obj); // Not an error
    #endif
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());

	// Free
	CUDA_CHECK_RETURN(hipFree(dev_comp_obj));
	assert(!myriad_dtor(comp_obj));

    hipDeviceReset();

	return EXIT_SUCCESS;
}

//////////////////////////////
// Test Device Symbol Malloc /
//////////////////////////////

__device__ float* my_float_ptr = NULL;

__global__ void cuda_dev_malloc_test()
{
	printf("my_float_ptr: %f\n", my_float_ptr[0]);
}

static int cuda_symbol_malloc()
{
	float* host_float_ptr = NULL, host_float_val = 5.0;
	CUDA_CHECK_RETURN(hipMalloc((void**)&host_float_ptr, sizeof(float)));
	CUDA_CHECK_RETURN(hipMemcpy(host_float_ptr, &host_float_val, sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(my_float_ptr), &host_float_ptr, sizeof(float*), size_t(0), hipMemcpyHostToDevice));
	
	    
    // BLAH
    const int nThreads = 1; // NUM_CUDA_THREADS;
    const int nBlocks = 1;

    dim3 dimGrid(nBlocks);
    dim3 dimBlock(nThreads);

    // Test
    #ifndef __clang__
    cuda_dev_malloc_test<<<dimGrid, dimBlock>>>(); // Not an error
    #endif
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());

    hipDeviceReset();

    return EXIT_SUCCESS;
}

///////////////////////
// HHCompartmentTest //
///////////////////////

__global__ void cuda_hh_compartment_test(void* hh_comp_obj)
{
	struct Compartment* self = (struct Compartment*) hh_comp_obj;
	struct CompartmentClass* self_c = (struct CompartmentClass*) cuda_myriad_class_of(self);
	printf("\tMy ptr: %p\n", self);
	printf("\tMy ID: %i\n", self->id);
	printf("\tMy class: %p\n", self->_.m_class);
	printf("\tGPU, my size: %lu\n", cuda_myriad_size_of(hh_comp_obj));
	printf("\tCompartment fxn: %p\n", self_c->m_comp_fxn);
	printf("\tCompartment fxn invocation: "); self_c->m_comp_fxn(self, NULL, 0.0, 0.0, 0);
	printf("\tCompartent fxn indirect call: "); cuda_simul_fxn(self, NULL, 0.0, 0.0, 0);
}

static int HHCompartmentTest()
{
	initCUDAObjects();
	initCompartment(1);
	initHHSomaCompartment(1);

	void* hh_comp_obj = NULL;
	void* dev_hh_comp_obj = NULL;

	hh_comp_obj = myriad_new(HHSomaCompartment, 5, 42, NULL, 50, NULL);

	simul_fxn(hh_comp_obj, NULL, 0.0, 0.0, 0);

	dev_hh_comp_obj = myriad_cudafy(hh_comp_obj, 0);
	
    // BLAH
    const int nThreads = 1; // NUM_CUDA_THREADS;
    const int nBlocks = 1;

    dim3 dimGrid(nBlocks);
    dim3 dimBlock(nThreads);

    // Test
    #ifndef __clang__
    cuda_hh_compartment_test<<<dimGrid, dimBlock>>>(dev_hh_comp_obj); // Not an error
    #endif
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());

	// Free
	assert(EXIT_SUCCESS == myriad_dtor(hh_comp_obj));


    hipDeviceReset();

    return EXIT_SUCCESS;
}

///////////////////
// Main function //
///////////////////
int main(int argc, char const *argv[])
{
    puts("Hello World!\n");

    UNIT_TEST_FUN(cuda_oop);
	UNIT_TEST_FUN(cuda_symbol_malloc);
	UNIT_TEST_FUN(mechanism_test);
	UNIT_TEST_FUN(compartment_test);
	UNIT_TEST_FUN(HHCompartmentTest);

    puts("\nDone.");

    return EXIT_SUCCESS;
}
